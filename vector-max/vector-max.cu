
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <unistd.h>
#include <stdlib.h>

/* gpuErrchk / gpuAssert copied from:
   https://stackoverflow.com/questions/14038589/what-is-the-canonical-way-to-check-for-errors-using-the-cuda-runtime-api .  */

#define gpuErrchk(ans)				\
  do {						\
    gpuAssert((ans), __FILE__, __LINE__);	\
  } while (0)

inline void
gpuAssert (hipError_t code, const char *file, int line, bool do_abort=true)
{
  if (code != hipSuccess)
    {
      fprintf (stderr,"GPUassert: %s %s %d\n",
	       hipGetErrorString (code), file, line);

      if (do_abort)
	abort ();
    }
}

__global__ void
hello (unsigned int *output)
{
  asm volatile ("{"

  ".reg .u32 r27;"
  ".reg .u64 rp;"
  ".reg .u32 r31;"
  ".reg .u32 r34;"
  ".reg .u32 r35;"
  ".reg .u32 r38;"
  ".reg .u32 r39;"
  ".reg .u32 r42;"
  ".reg .u32 r43;"
  ".reg .u32 r46;"
  ".reg .u32 r47;"
  ".reg .u32 r50;"
  ".reg .u32 r51;"
  ".reg .pred r63;"
  ".reg .pred r66;"
  ".reg .u32 r71;"
  ".reg .pred r77;"
  ".reg .pred r80;"
  ".reg .u32 r81;"
  ".reg .pred r82;"
  ".reg .u32 r83;"

  ".reg .u32 %x;"
  "mov.u32 %x,%tid.x;"
  "setp.ne.u32 r77,%x,0;"

  "mov.u64 rp, %0;"

  "@ r77 bra $L17;"
  "mov.u32 r71,1;"
 "$L17:"

  "bra $L4;"

 "$L7:"
  // Unreachable.
  "trap;"

 "$L4:"
  "@ r77 bra $L16;"
  "mov.u32 r27,r71;"
 "$L16:"

  "bra $L2;"

 "$L6:"
  // Unreachable.
  "trap;"
  
 "$L2:"
  "shfl.sync.idx.b32 r27,r27,0,31,0xffffffff;"
  "shfl.sync.idx.b32 r71,r71,0,31,0xffffffff;"

  "mov.u32 r31,%tid.x;"

  "shfl.sync.down.b32 r34,r31,16,31,0xffffffff;"
  "max.s32 r35,r34,r31;"

  "shfl.sync.down.b32 r38,r35,8,31,0xffffffff;"
  "max.s32 r39,r38,r35;"

  "shfl.sync.down.b32 r42,r39,4,31,0xffffffff;"
  "max.s32 r43,r42,r39;"

  "shfl.sync.down.b32 r46,r43,2,31,0xffffffff;"
  "max.s32 r47,r46,r43;"

  "shfl.sync.down.b32 r50,r47,1,31,0xffffffff;"

  "setp.eq.u32 r82,1,0;"

  "@ r77 bra $L14;"
  "setp.ne.u32 r63,r27,1;"
  "mov.pred r82,r63;"
 "$L14:"
  
  "mov.pred r63,r82;"
  "selp.u32 r83,1,0,r63;"
  "shfl.sync.idx.b32 r83,r83,0,31,0xffffffff;"
  "setp.ne.u32 r63,r83,0;"
  "@ r63 bra.uni $L6;"

  "setp.eq.u32 r80,1,0;"

  "@ r77 bra $L13;"
  "max.s32 r51,r50,r47;"
 "$L13:"

  "mov.pred r66,r80;"
  "selp.u32 r81,1,0,r66;"
  "shfl.sync.idx.b32 r81,r81,0,31,0xffffffff;"
  "setp.ne.u32 r66,r81,0;"
  "@ r66 bra.uni $L7;"

  "@ r77 bra $L100;"
  "st.u32 [rp], r51;"
 "$L100:"

  "}" : : "l"(output));
}

#define BSIZE 1
unsigned int a[BSIZE];

int
main (void)
{
  /* Dimensions: just one warp.  */
  #define WARP_SIZE 32
  #define NR_WARPS 1
  dim3 dimBlock (WARP_SIZE, NR_WARPS);
  dim3 dimGrid (1, 1);

  /* Initialize a.  */
  for (int i = 0; i < BSIZE; ++i)
    a[i] = 0;

  /* Allocate device copy of a.  */
  unsigned int *p;
  gpuErrchk ((hipMalloc ((void**)&p, BSIZE * sizeof(int))));

  /* Copy to device.  */
  gpuErrchk ((hipMemcpy (p, &a[0], BSIZE * sizeof (int), hipMemcpyHostToDevice)));

  /* Execute kernel.  */
  hello<<<dimGrid, dimBlock>>> (p);

  /* Copy back to host.  */
  gpuErrchk ((hipMemcpy (&a[0], p, BSIZE * sizeof (int), hipMemcpyDeviceToHost)));

  /* Print output.  */
  for (int i = 0; i < BSIZE; ++i)
    printf ("a[%d]: %u\n", i, a[i]);

  if (a[0] != 31)
    __builtin_abort ();  

  return 0;
}
