
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <unistd.h>
#include <stdlib.h>

__global__ void
hello (unsigned int *output)
{
  asm volatile ("{"

  ".reg .u32 r27;"
  ".reg .u64 rp;"
  ".reg .u32 r31;"
  ".reg .u32 r34;"
  ".reg .u32 r35;"
  ".reg .u32 r38;"
  ".reg .u32 r39;"
  ".reg .u32 r42;"
  ".reg .u32 r43;"
  ".reg .u32 r46;"
  ".reg .u32 r47;"
  ".reg .u32 r50;"
  ".reg .u32 r51;"
  ".reg .pred r63;"
  ".reg .pred r66;"
  ".reg .u32 r71;"
  ".reg .pred r77;"
  ".reg .pred r80;"
  ".reg .u32 r81;"
  ".reg .pred r82;"
  ".reg .u32 r83;"

  ".reg .u32 %x;"
  "mov.u32 %x,%tid.x;"
  "setp.ne.u32 r77,%x,0;"

  "mov.u64 rp, %0;"

  "@ r77 bra $L17;"
  "mov.u32 r71,1;"
 "$L17:"

  "bra $L4;"

 "$L7:"
  // Unreachable.
  "trap;"

 "$L4:"
  "@ r77 bra $L16;"
  "mov.u32 r27,r71;"
 "$L16:"

  "bra $L2;"

 "$L6:"
  // Unreachable.
  "trap;"
  
 "$L2:"
  "shfl.sync.idx.b32 r27,r27,0,31,0xffffffff;"
  "shfl.sync.idx.b32 r71,r71,0,31,0xffffffff;"

  "mov.u32 r31,%tid.x;"

  "shfl.sync.down.b32 r34,r31,16,31,0xffffffff;"
  "max.s32 r35,r34,r31;"

  "shfl.sync.down.b32 r38,r35,8,31,0xffffffff;"
  "max.s32 r39,r38,r35;"

  "shfl.sync.down.b32 r42,r39,4,31,0xffffffff;"
  "max.s32 r43,r42,r39;"

  "shfl.sync.down.b32 r46,r43,2,31,0xffffffff;"
  "max.s32 r47,r46,r43;"

  "shfl.sync.down.b32 r50,r47,1,31,0xffffffff;"

  "setp.eq.u32 r82,1,0;"

  "@ r77 bra $L14;"
  "setp.ne.u32 r63,r27,1;"
  "mov.pred r82,r63;"
 "$L14:"
  
  "mov.pred r63,r82;"
  "selp.u32 r83,1,0,r63;"
  "shfl.sync.idx.b32 r83,r83,0,31,0xffffffff;"
  "setp.ne.u32 r63,r83,0;"
  "@ r63 bra.uni $L6;"

  "setp.eq.u32 r80,1,0;"

  "@ r77 bra $L13;"
  "max.s32 r51,r50,r47;"
 "$L13:"

  "mov.pred r66,r80;"
  "selp.u32 r81,1,0,r66;"
  "shfl.sync.idx.b32 r81,r81,0,31,0xffffffff;"
  "setp.ne.u32 r66,r81,0;"
  "@ r66 bra.uni $L7;"

  "@ r77 bra $L100;"
  "st.u32 [rp], r51;"
 "$L100:"

  "}" : : "l"(output));
}

#define BSIZE 1
unsigned int a[BSIZE];

int
main (void)
{
  hipError_t res;

  /* Dimensions: just one warp.  */
  #define WARP_SIZE 32
  #define NR_WARPS 1
  dim3 dimBlock (WARP_SIZE, NR_WARPS);
  dim3 dimGrid (1, 1);

  /* Initialize a.  */
  for (int i = 0; i < BSIZE; ++i)
    a[i] = 0;

  /* Allocate device copy of a.  */
  unsigned int *p;
  res = hipMalloc ((void**)&p, BSIZE * sizeof(int)); 
  if (res != hipSuccess)
    abort ();

  /* Copy to device.  */
  res = hipMemcpy (p, &a[0], BSIZE * sizeof (int), hipMemcpyHostToDevice); 
  if (res != hipSuccess)
    abort ();

  /* Execute kernel.  */
  hello<<<dimGrid, dimBlock>>> (p);

  /* Copy back to host.  */
  res = hipMemcpy (&a[0], p, BSIZE * sizeof (int), hipMemcpyDeviceToHost);
  if (res != hipSuccess)
    abort ();

  /* Print output.  */
  for (int i = 0; i < BSIZE; ++i)
    printf ("a[%d]: %u\n", i, a[i]);

  if (a[0] != 31)
    __builtin_abort ();  

  return 0;
}
