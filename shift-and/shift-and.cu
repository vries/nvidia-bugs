
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <unistd.h>
#include <stdlib.h>

#define gpuErrchk(ans)				\
  do {						\
    gpuAssert((ans), __FILE__, __LINE__);	\
  } while (0)

inline void
gpuAssert (hipError_t code, const char *file, int line)
{
  if (code != hipSuccess)
    {
      fprintf (stderr,"GPUassert: %s %s %d\n",
	       hipGetErrorString (code), file, line);

      abort ();
    }
}

__global__ void
hello (unsigned long long d0, unsigned long long int *output)
{
  asm volatile
    (
     ".reg .u64 ar0;"
     "mov.u64 ar0, %0;"

     ".reg .u64 rp;"
     "mov.u64 rp, %1;"

     ".local .align 8 .b8 stack_ar[16];"
     ".reg .u64 stack;"
     "cvta.local.u64 stack,stack_ar;"

     ".reg .u64 r24;"
     ".reg .u64 r27;"
     ".reg .u64 r28;"
     ".reg .pred r29;"
     ".reg .u32 r30;"
     ".reg .u64 r31;"
     ".reg .u64 r32;"
     ".reg .pred r33;"
     ".reg .pred r34;"

     "mov.u64 r27,ar0;"
     "shr.u64 r28,r27,56;"
     "setp.ne.u64 r29,r28,0;"
     "@ r29 bra $L5;"

     "mov.u64 r24,56;"
     "bra $L3;"

     "$L4:"
     "cvt.u32.u64 r30,r24;"
     "shr.u64 r31,r27,r30;"
     "and.b64 r32,r31,255;"
     "setp.ne.u64 r33,r32,0;"
     "@ r33 bra $L2;"

     "$L3:"
     "add.u64 r24,r24,-8;"
     "setp.ne.u64 r34,r24,0;"
     "@ r34 bra $L4;"
     "bra $L2;"

     "$L5:"
     "mov.u64 r24,56;"

     "$L2:"
     "st.u64 [stack],r24;"
     "ld.u64 r24,[stack];"
     "st.u64 [rp],r24;"
     ::"l"(d0), "l"(output));
}

#define BSIZE 1
unsigned long long int a[BSIZE];

int
main (void)
{
  dim3 dimBlock (1, 1);
  dim3 dimGrid (1, 1);

  /* Initialize a.  */
  for (int i = 0; i < BSIZE; ++i)
    a[i] = 0;

  /* Allocate device copy of a.  */
  unsigned long long int *p;
  gpuErrchk ((hipMalloc ((void**)&p, BSIZE * sizeof(*a))));

  /* Copy to device.  */
  gpuErrchk ((hipMemcpy (p, &a[0], BSIZE * sizeof (*a), hipMemcpyHostToDevice)));

  /* Execute kernel.  */
  hello<<<dimGrid, dimBlock>>> (1, p);

  /* Copy back to host.  */
  gpuErrchk ((hipMemcpy (&a[0], p, BSIZE * sizeof (*a), hipMemcpyDeviceToHost)));

  /* Print output.  */
  for (int i = 0; i < BSIZE; ++i)
    printf ("a[%d]: %llx\n", i, a[i]);

  return 0;
}
