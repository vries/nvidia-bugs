#define DEBUG 1


#include <hip/hip_runtime.h>
#include <assert.h>
#if DEBUG
#include <stdio.h>
#endif

#define gpuErrchk(ans)				\
  do {						\
    gpuAssert ((ans), __FILE__, __LINE__);	\
  } while (0)

inline void
gpuAssert (hipError_t code, const char *file, int line)
{
  if (code != hipSuccess)
    {
#if DEBUG
      fprintf (stderr, "GPUassert: %s %s %d\n", hipGetErrorString (code),
	       file, line);
#endif
    }

  assert (code == hipSuccess);
}

__device__ int
foo (int n)
{
  int res;
  
  asm volatile ("{"

		".reg .u32 r25;"
		".reg .u64 r26;"
		".reg .u32 r27;"
		".reg .u32 r28;"
		".reg .u32 r31;"

		"mov.u32 r27, %1;"

		"mov.u32 r28,-2147483648;"

		"mul.wide.s32 r26,r27,r28;"

		"set.eq.u32.u64 r31,r26,2147483648;"

		"neg.s32 r25,r31;"

		"mov.u32 %0, r25;"

		"}"

		: "=r"(res) : "r"(n));

  return res;
}

__global__  void
hello (int *p)
{
  *p = foo (*p);
}

int
main (void)
{
  dim3 dimBlock (1, 1);
  dim3 dimGrid (1, 1);

  int *p;
  int n = -1;

  gpuErrchk ( hipMalloc ((void**)&p, sizeof(int)) );
  gpuErrchk ( hipMemcpy (p, &n, sizeof (int), hipMemcpyHostToDevice) );

  hello<<<dimGrid, dimBlock>>> (p);
  gpuErrchk( hipPeekAtLastError() );
  gpuErrchk( hipDeviceSynchronize() );

  gpuErrchk ( hipMemcpy (&n, p, sizeof (int), hipMemcpyDeviceToHost) );

#if DEBUG
  if (n == 1)
    printf ("n: %d (GOOD)\n", n);
  else
    printf ("n: %d (BAD)\n", n);  
#endif

  assert (n == 1);

  return 0;
}
