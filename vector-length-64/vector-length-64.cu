
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <unistd.h>
#include <stdlib.h>

/* gpuErrchk / gpuAssert copied from:
   https://stackoverflow.com/questions/14038589/what-is-the-canonical-way-to-check-for-errors-using-the-cuda-runtime-api .  */

#define gpuErrchk(ans)				\
  do {						\
    gpuAssert((ans), __FILE__, __LINE__);	\
  } while (0)

inline void
gpuAssert (hipError_t code, const char *file, int line, bool do_abort=true)
{
  if (code != hipSuccess)
    {
      fprintf (stderr,"GPUassert: %s %s %d\n",
	       hipGetErrorString (code), file, line);

      if (do_abort)
	abort ();
    }
}


__global__ void
hello ()
{
  asm volatile ("{"

  ".shared .align 8 .u8 __oacc_bcast[144];"

  ".local .align 16 .b8 %frame_ar[24];"
  ".reg .u64 %frame;"
  "cvta.local.u64 %frame,%frame_ar;"

  ".reg .u32 %r22;"
  ".reg .u32 %r23;"
  ".reg .u32 %r24;"
  ".reg .u32 %r25;"
  ".reg .u32 %r28;"
  ".reg .u32 %r29;"
  ".reg .u32 %r30;"
  ".reg .u32 %r31;"
  ".reg .u32 %r34;"
  ".reg .u32 %r35;"
  ".reg .u32 %r36;"
  ".reg .u32 %r37;"
  ".reg .u32 %r38;"
  ".reg .u32 %r39;"
  ".reg .u64 %r40;"
  ".reg .u64 %r43;"
  ".reg .pred %r44;"
  ".reg .pred %r45;"
  ".reg .u32 %r46;"
  ".reg .u32 %r47;"
  ".reg .u32 %r48;"
  ".reg .u64 %r49;"
  ".reg .u64 %r50;"
  ".reg .u64 %r51;"
  ".reg .u64 %r52;"
  ".reg .u32 %r53;"
  ".reg .pred %r54;"
  ".reg .pred %r55;"
  ".reg .u32 %r56;"
  ".reg .pred %r57;"
  ".reg .pred %r58;"
  ".reg .u32 %r59;"
  ".reg .u64 %r60;"
  ".reg .u64 %r61;"
  ".reg .u64 %r62;"
  ".reg .u32 %r63;"
  ".reg .pred %r64;"
  ".reg .u64 %r65;"
  ".reg .u64 %r66;"
  ".reg .u32 %r67;"
  ".reg .u64 %r68;"
  ".reg .u64 %r69;"
  ".reg .u64 %r70;"
  ".reg .u32 %r71;"
  ".reg .pred %r72;"
  ".reg .u64 %r73;"
  ".reg .u64 %r74;"
  ".reg .u64 %r75;"
  ".reg .u64 %r76;"
  ".reg .u32 %r77;"
  ".reg .pred %r78;"
  ".reg .u64 %r79;"
  ".reg .u64 %r80;"
  ".reg .u64 %r81;"
  ".reg .u64 %r82;"
  ".reg .u32 %r83;"
  ".reg .pred %r84;"
  ".reg .u64 %r85;"
  ".reg .pred %r86;"
  ".reg .u32 %r87;"
  ".reg .u32 %r88;"
  ".reg .u32 %r89;"
  ".reg .u32 %r90;"
  ".reg .u32 %r91;"
  ".reg .u32 %r92;"
  ".reg .pred %r93;"

  "{"
   ".reg .u32 %y;"
   " mov.u32 %y,%tid.y;"
   " setp.ne.u32 %r93,%y,0;"
  "}"

  "{"
    ".reg .u32 %x;"
    "mov.u32 %x,%tid.x;"
    "setp.ne.u32 %r86,%x,0;"
  "}"

  "{"
    ".reg .u32 %tidy;"
    ".reg .u64 %t_bcast;"
    ".reg .u64 %y64;"
    "mov.u32 %tidy,%tid.y;"
    "cvt.u64.u32 %y64,%tidy;"
    "add.u64 %y64,%y64,1;"
    // vector ID
    "cvta.shared.u64 %t_bcast,__oacc_bcast;"
    "mad.lo.u64 %r66,%y64,48,%t_bcast;"
    // vector broadcast offset
    "add.u32 %r67,%tidy,1;"
    // vector synchronization barrier
  "}"

  "@ %r93 bra.uni $L18;"
  "@ %r86 bra $L19;"
  "st.u64 [%frame],0;"
  // fork 2;"
  "cvta.shared.u64 %r85,__oacc_bcast;"
  "mov.u64 %r82,%frame;"
  "mov.u32 %r83,1;"
 "$L11:"
  "sub.u32 %r83,%r83,1;"
  "ld.u64 %r81,[%r82];"
  "st.u64 [%r85],%r81;"
  "add.u64 %r85,%r85,8;"
  "setp.ne.u32 %r84,%r83,0;"
  "add.u64 %r82,%r82,8;"
  "@ %r84 bra $L11;"
 "$L19:"
 "$L18:"

  "barrier.sync.aligned 0;"

  "@ %r86 bra $L12;"
  "cvta.shared.u64 %r79,__oacc_bcast;"
  "mov.u64 %r76,%frame;"
  "mov.u32 %r77,1;"
 "$L10:"
  "sub.u32 %r77,%r77,1;"
  "ld.u64 %r75,[%r79];"
  "add.u64 %r79,%r79,8;"
  "st.u64 [%r76],%r75;"
  "setp.ne.u32 %r78,%r77,0;"
  "add.u64 %r76,%r76,8;"
  "@ %r78 bra $L10;"
 "$L12:"

 "$L7:"

  "@ %r86 bra $L13;"
  "st.u32 [%r66],0;"
 "$L13:"

  "barrier.sync %r67,64;"

  "@ %r86 bra $L16;"
  "st.u32 [%r66],0;"
 "$L16:"

  "barrier.sync %r67,64;"

  "ld.u32 %r92,[%r66];"
  "setp.ne.u32 %r58,%r92,0;"

  "barrier.sync %r67,64;"

  "@ %r58 bra.uni $L7;"

  "}" : :);
}

int
main (void)
{
  #define WARP_SIZE 32
  dim3 dimBlock (WARP_SIZE * 2, 2);
  dim3 dimGrid (1, 1);

  hello<<<dimGrid, dimBlock>>> ();

  gpuErrchk (hipDeviceSynchronize ());

  return 0;
}
