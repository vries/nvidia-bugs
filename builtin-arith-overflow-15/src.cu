
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <unistd.h>
#include <stdlib.h>

#define gpuErrchk(ans)				\
  do {						\
    gpuAssert((ans), __FILE__, __LINE__);	\
  } while (0)

inline void
gpuAssert (hipError_t code, const char *file, int line, bool do_abort=true)
{
  if (code != hipSuccess)
    {
      fprintf (stderr,"GPUassert: %s %s %d\n",
	       hipGetErrorString (code), file, line);

      if (do_abort)
	abort ();
    }
}

__global__ void
hello (unsigned int *output)
{
  asm volatile (
    "{"

    ".reg .u64 rp;"
    "mov.u64 rp, %0;"

    ".local .u16 frame_var;"

    ".reg .u16 r22;"
    ".reg .u16 r32;"
    ".reg .u16 r33;"
    ".reg .u32 r35;"

    "mov.u16 r22, 0x0080;"

    "st.local.u16 [frame_var],r22;"
    "ld.local.u16 r32,[frame_var];"
    //"mov.u16 r32,0x0080;"

    "sub.u16 r33,0x0000,r32;"
    //"mov.u16 r33,0xff80;"

    "cvt.u32.u16 r35,r33;"
    //"mov.u32 r35, 0x0000ff80;"

    "st.u32 [rp], r35;"
  "}" : : "l"(output));
}

#define BSIZE 1
unsigned int a[BSIZE];

int
main (void)
{
  /* Dimensions: just one thread.  */
  dim3 dimBlock (1, 1);
  dim3 dimGrid (1, 1);

  /* Initialize a.  */
  for (int i = 0; i < BSIZE; ++i)
    a[i] = 0;

  /* Allocate device copy of a.	 */
  unsigned int *p;
  gpuErrchk ((hipMalloc ((void**)&p, BSIZE * sizeof(int))));

  /* Copy to device.  */
  gpuErrchk ((hipMemcpy (p, &a[0], BSIZE * sizeof (int),
			  hipMemcpyHostToDevice)));

  /* Execute kernel.  */
  hello<<<dimGrid, dimBlock>>> (p);

  /* Copy back to host.	 */
  gpuErrchk ((hipMemcpy (&a[0], p, BSIZE * sizeof (int),
			  hipMemcpyDeviceToHost)));

  /* Print output.  */
  for (int i = 0; i < BSIZE; ++i)
    printf ("a[%d]: %x\n", i, a[i]);

  if (a[0] != 0x0000ff80)
    __builtin_abort ();

  return 0;
}
