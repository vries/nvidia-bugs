
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <unistd.h>
#include <stdlib.h>

#define gpuErrchk(ans)				\
  do {						\
    gpuAssert((ans), __FILE__, __LINE__);	\
  } while (0)

inline void
gpuAssert (hipError_t code, const char *file, int line)
{
  if (code != hipSuccess)
    {
      fprintf (stderr,"GPUassert: %s %s %d\n",
	       hipGetErrorString (code), file, line);

      abort ();
    }
}

__global__ void
hello (unsigned int *output)
{
  asm volatile ("{"

  ".reg .u64 rp;"
  "mov.u64 rp, %0;"

  ".reg .u32 r22;"
  ".reg .u32 r24;"
  ".reg .u32 r27;"
  ".reg .u32 r30;"
  ".reg .u32 r31;"
  ".reg .u32 r34;"
  ".reg .u32 r35;"
  ".reg .u32 r38;"
  ".reg .u32 r39;"
  ".reg .u32 r42;"
  ".reg .u32 r43;"
  ".reg .u32 r46;"
  ".reg .u64 r48;"
  ".reg .pred r49;"
  ".reg .u32 r51;"
  ".reg .pred r52;"
  ".reg .u64 r53;"
  ".reg .pred r59;"
  ".reg .pred r60;"
  ".reg .u32 r61;"
  ".reg .pred r62;"
  ".reg .u32 r63;"

  "{"
    ".reg .u32 %x;"
    "mov.u32 %x,%tid.x;"
    "setp.ne.u32 r59,%x,0;"
  "}"

  "@ r59 bra $L15;"
  "mov.u32 r22,2;" // Initialize outer loop counter.
 "$L15:"
  
  "bra $L3;" // Goto inner loop start.

 "$L6:" // Outer loop backedge target.

  "@ r59 bra $L10;"
  "mov.u32 r22,1;" // Increment outer loop counter.
 "$L10:"

 "$L3:" // Inner loop start.

  "@ r59 bra $L14;"
  "mov.u32 r24,2;" // Initialize inner loop counter.
 "$L14:"

  "bra $L2;"

 "$L7:" // Inner loop backedge target.

  "@ r59 bra $L13;"
  "mov.u32 r24,1;" // Increment inner loop counter.
 "$L13:"

  // Loop body.
 "$L2:"
  "mov.u32 r27,%tid.x;"
  "shfl.down.b32 r30,r27,16,31;"
  "max.s32 r31,r30,r27;"
  "shfl.down.b32 r34,r31,8,31;"
  "max.s32 r35,r34,r31;"
  "shfl.down.b32 r38,r35,4,31;"
  "max.s32 r39,r38,r35;"
  "shfl.down.b32 r42,r39,2,31;"
  "max.s32 r43,r42,r39;"
  "shfl.down.b32 r46,r43,1,31;"
  "max.s32 r51,r46,r43;"
  // Assert: r51 == 31.

  "setp.eq.u32 r62,1,0;" // Initialize predicate for all lanes.

  "@ r59 bra $L12;"
  "st.u32 [rp],r51;" // Result store.
  "setp.ne.u32 r62,r24,1;" // Calculate inner loop condition.
 "$L12:"

  "mov.pred r52,r62;"
  "selp.u32 r63,1,0,r52;"
  "shfl.idx.b32 r63,r63,0,31;" // Broadcast inner loop condition.
  "setp.ne.u32 r52,r63,0;"

  "@ r52 bra.uni $L7;" // Continue inner loop.

  "setp.eq.u32 r60,1,0;" // Initialize predicate for all lanes.

  "@ r59 bra $L11;"
  "setp.ne.u32 r60,r22,1;" // Calculate outer loop condition.
 "$L11:"

  "mov.pred r49,r60;"
  "selp.u32 r61,1,0,r49;"
  "shfl.idx.b32 r61,r61,0,31;" // Broadcast outer loop condition.
  "setp.ne.u32 r49,r61,0;"
  "@ r49 bra.uni $L6;" // Continue outer loop.

  "}" : : "l"(output));
}

#define BSIZE 1
unsigned int a[BSIZE];

int
main (void)
{
  /* Dimensions: just one warp.  */
  #define WARP_SIZE 32
  #define NR_WARPS 1
  dim3 dimBlock (WARP_SIZE, NR_WARPS);
  dim3 dimGrid (1, 1);

  /* Initialize a.  */
  for (int i = 0; i < BSIZE; ++i)
    a[i] = 0;

  /* Allocate device copy of a.  */
  unsigned int *p;
  gpuErrchk ((hipMalloc ((void**)&p, BSIZE * sizeof(int))));

  /* Copy to device.  */
  gpuErrchk ((hipMemcpy (p, &a[0], BSIZE * sizeof (int), hipMemcpyHostToDevice)));

  /* Execute kernel.  */
  hello<<<dimGrid, dimBlock>>> (p);

  /* Copy back to host.  */
  gpuErrchk ((hipMemcpy (&a[0], p, BSIZE * sizeof (int), hipMemcpyDeviceToHost)));

  /* Print output.  */
  for (int i = 0; i < BSIZE; ++i)
    printf ("a[%d]: %u\n", i, a[i]);

  if (a[0] != 31)
    __builtin_abort ();  

  return 0;
}
